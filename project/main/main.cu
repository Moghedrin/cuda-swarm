#include "hip/hip_runtime.h"
//#include <hip/hip_runtime.h>
//#include <stdio.h>
#include "../wrapper_classes/Swarm.h"
/*
#include "../pso/pso.h"
dim3 threadGrids(128, 128, 1); // MAX NUMBER OF BLOCKS : 65535 ^ 3
dim3 threadBlocks(512, 1, 1); // MAX NUMBER OF THREADS : 512

bool swapper = true;
blockData * hostData = NULL;
blockData * deviData = NULL;
unsigned int blockSize = sizeof(blockData);

const unsigned int SEED = 42;
const float MININIT = -100.0;
const float MAXINIT = 100.0;
const float MINVEL = -10.0;
const float MAXVEL = 10.0;

void printOutHostData() {
	for(unsigned int i = 0; i < PARTICLE_COUNT; i++) {
		printf("<particle Source %d>\n", i);
		printf("<particle Destination %d>\n", i);
		for(unsigned int j = 0; j < DIM; j++) {
			printf("\t<dim Source %d: %.2f>\n", j, hostData->s[i].pos[j]);
			printf("\t<dim Destination %d: %.2f>\n", j, hostData->d[i].pos[j]);
			printf("\t<bestDim Source %d: %.2f>\n", j, hostData->s[i].bsf[j]);
			printf("\t<bestDim Destination%d: %.2f>\n", j, hostData->d[i].bsf[j]);
			printf("\t<delta Source %d: %.2f>\n", j, hostData->s[i].del[j]);
			printf("\t<delta Destination %d: %.2f>\n", j, hostData->d[i].del[j]);
		}
	}
}

void runSingle(float3 * pos) {
	pso<<<threadGrids, threadBlocks>>>(deviData, swapper);
	swapper = !swapper;
	hipMemcpy(hostData, deviData, blockSize, hipMemcpyDeviceToHost);
	printOutHostData();
}

void initialize() {
	hostData = (blockData *)malloc(blockSize);
	memset(hostData, 0, blockSize);
	hipMalloc((void **)&deviData, blockSize);
	//CudaMemcpy(void * dest, void * src, size, direction)
	hipMemcpy(deviData, hostData, blockSize, hipMemcpyHostToDevice);
	initBlock<<<threadGrids, threadBlocks>>>(deviData, SEED, 10.0, MININIT, MAXINIT, MINVEL, MAXVEL);
	hipMemcpy(hostData, deviData, blockSize, hipMemcpyDeviceToHost);
	printOutHostData();
}

void finalize() {
	free(hostData);
	hipFree(deviData);
}

void runPSO(unsigned int iterations) {
	for(unsigned int i = 0; i < iterations; i++) {
		pso<<<threadGrids, threadBlocks>>>(deviData, swapper);
		swapper = !swapper;
	}
}

void copyResultsBack() {
	hipMemcpy(hostData, deviData, blockSize, hipMemcpyDeviceToHost);
}
*/
__device__ float f1(float * a) {
	float n = 0.0;
	for(unsigned int i = 0; i < DIM; i++) {
		n += a[i] * a[i];
	}
	return (1.0/(sqrtf(n)+1)) * 100.0;
}

__device__ fold k = f1;

int main(int argc, char ** argv) {
	Swarm<fold> a(43, 10.0, -100.0, 100.0, -10.0, 10.0, k);
	a.printCurrent();
	a.runNIterations(10000);
	a.printCurrent();
	//initialize();
	//printOutHostData();
	return 0;
}
