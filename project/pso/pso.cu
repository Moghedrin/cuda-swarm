#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "pso.h"
#include "../headers/max_func.h"

__device__ void src_to_dest(particle * s, particle * d, unsigned int index, unsigned int dim) {
	d[index].pos[dim] = s[index].pos[dim];
	d[index].del[dim] = 0.0;
	d[index].bsf[dim] = s[index].bsf[dim];
	d[index].max_v = s[index].max_v;
}

__device__ unsigned int global(particle * s, unsigned int i) {
	unsigned int pu, pd;
	float am, bm, cm;

	pu = (i + 1) % DIM;
	pd = (i - 1) % DIM;
	am = max_func(s[i].bsf);
	bm = max_func(s[pu].bsf);
	cm = max_func(s[pd].bsf);
	
	if (am > bm) {
		if (am > cm) {
			return i;
		}
		else {
			return pd;
		}
	}
	else {
		if (bm > cm) {
			return pu;
		}
		else {
			return pd;
		}
	}
}

__device__ float inertial(float del) {
	return W * del;
}

__device__ float cognitive(float pos, float bsf) {
	return C1 * (bsf - pos);
}

__device__ float social(particle * s, particle * d, unsigned int i, unsigned int dim) {
	unsigned int best_index = global(s, i);
	return C2 * (s[best_index].bsf[dim] - s[i].pos[dim]);
}

__device__ void update_best(particle * s, particle * d, unsigned int index) {
	if (max_func(s[index].pos) > max_func(s[index].bsf)) {
		for(unsigned int i = 0; i < DIM; i++) {
			d[index].bsf[i] = s[index].pos[i];
		}
	}
}

__device__ void update(particle * s, particle * d, hiprandState_t * state, unsigned int index) {
	for(unsigned int i = 0; i < DIM; i++) {
		src_to_dest(s, d, index, i);
		d[index].del[i] += inertial(s[index].del[i]);
		d[index].del[i] += cognitive(s[index].pos[i], s[index].bsf[i]) * hiprand_uniform(&state[index]);
		d[index].del[i] += social(s, d, index, i) * hiprand_uniform(&state[index]);
		
		float delp = d[index].del[i];
		if(abs(d[index].del[i]) > delp) {
			delp = d[index].max_v;
			if(d[index].del[i] < 0) {
				delp *= -1;
			}
			d[index].del[i] = delp;
		}
		
		d[index].pos[i] += delp;

	}
}

__global__ void initBlock(blockData * p, unsigned int seed, float max_v, float pos_min, float pos_max, float del_min, float del_max) {
	unsigned int x_i = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int y_i = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned i = x_i + y_i * blockDim.x * gridDim.x;
	
	if (i < PARTICLE_COUNT) {
		hiprand_init(seed, i, 0, &(p->states[i]));
		for (unsigned int j = 0; j < DIM; j++) {
			float k = (pos_max - pos_min) * hiprand_uniform(&(p->states[i])) + pos_min;
			p->s[i].pos[j] = k;
			p->s[i].bsf[j] = k;
			p->s[i].del[j] = (del_max - del_min) * hiprand_uniform(&(p->states[i])) + del_min;
			p->s[i].max_v = max_v;
		}
	}
}

__global__ void pso(blockData * p, bool sw) {
	unsigned int x_i = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int y_i = threadIdx.y + blockIdx.y * blockDim.y;
	unsigned i = x_i + y_i * blockDim.x * gridDim.x;
	
	particle * s = sw ? (particle *)p->s : (particle *)p->d;
	particle * d = sw ? (particle *)p->d : (particle *)p->s;
	hiprandState_t * states = (hiprandState_t *)p->states;
	
	if (i < PARTICLE_COUNT) {
		update(s, d, states, i);
		update_best(s, d, i);
	}
}
